#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <chrono>

// Matrix dimensions
const int nb_genes = 10000; // number of gens (columns)
const int nb_samples = 80; // number of samples (rows)


int threads_per_block = 128;
int num_pairs = (nb_genes * (nb_genes - 1)) / 2;
int num_blocks = num_pairs;

#define BLOCK_SIZE 128

__global__
void computeLogRatioVariance(float *d_Y, float *d_variances, int nb_samples, int nb_genes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nb_genes && j < i) {
        // Pack accumulators together to encourage fusion
        float2 accum = make_float2(0.0f, 0.0f);
        int k = 0;

        // Process 4 samples at a time with vector loads
        #pragma unroll
        for (; k <= nb_samples - 4; k += 4) {
            float4 y_i = *reinterpret_cast<float4*>(&d_Y[k + i * nb_samples]);
            float4 y_j = *reinterpret_cast<float4*>(&d_Y[k + j * nb_samples]);

            // Use intrinsics that compiler can fuse
            #pragma unroll
            for (int m = 0; m < 4; ++m) {
                // __fdividef has lower precision but can be fused
                float ratio = __fdividef((&y_i.x)[m], (&y_j.x)[m]);
                // __logf can be fused with multiply/add operations
                float log_val = __logf(ratio);

                // Accumulate sum and square together
                accum.x = __fmaf_rn(1.0f, log_val, accum.x); // sum += log_val
                accum.y = __fmaf_rn(log_val, log_val, accum.y); // sumsq += log_val * log_val
            }
        }

        // Handle remaining elements with same fused operations
        for (; k < nb_samples; ++k) {
            float yi = d_Y[k + i * nb_samples];
            float yj = d_Y[k + j * nb_samples];

            float ratio = __fdividef(yi, yj);
            float log_val = __logf(ratio);

            accum.x = __fmaf_rn(1.0f, log_val, accum.x);
            accum.y = __fmaf_rn(log_val, log_val, accum.y);
        }

        // Fused mean and variance computation
        float inv_n = __frcp_rn(static_cast<float>(nb_samples));
        float mean = accum.x * inv_n;
        float variance = (accum.y - __fmul_rn(nb_samples, __fmul_rn(mean, mean))) * __frcp_rn(static_cast<float>(nb_samples - 1));

        int pair_index = (i * (i - 1)) / 2 + j;
        d_variances[pair_index] = variance;
    }
}

// CPU implementation for log variance ratio benchmark
float* compute_log_variance_ratio_cpu(const float* Y, int nb_samples, int nb_genes) {
    // Output array to store variances for each pair
    int num_pairs = (nb_genes * (nb_genes - 1)) / 2;
    float* variances = new float[num_pairs];
    int counter = 0;

    // For each pair of genes
    for(int i = 1; i < nb_genes; i++) {
        for(int j = 0; j < i; j++) {
            float mean = 0.0f;
            float variance = 0.0f;
            
            // First pass: compute mean of log ratios
            for(int k = 0; k < nb_samples; k++) {
                float ratio = Y[k + i * nb_samples] / Y[k + j * nb_samples];
                mean += log(ratio);
            }
            mean /= nb_samples;
            
            // Second pass: compute variance
            for(int k = 0; k < nb_samples; k++) {
                float ratio = Y[k + i * nb_samples] / Y[k + j * nb_samples];
                float diff = log(ratio) - mean;
                variance += diff * diff;
            }
            
            // Divide by (N-1) for sample variance
            variances[counter] = variance / (nb_samples - 1);
            counter++;
        }
    }

    return variances;
}

struct PerformanceMetrics {
    float kernel_time;      // milliseconds
    float memory_time;      // milliseconds
    float total_time;       // milliseconds
    float gflops;          // Floating point operations per second
    float bandwidth;       // GB/s
};

void initializeMatrice(float* Y, int nb_samples, int nb_genes) {
    for(int i = 0; i < nb_samples * nb_genes; i++) Y[i] = rand() / (float)RAND_MAX;
}

bool verifyResults(float* variances_gpu, float* variances_cpu, int num_pairs) {
    const float epsilon = 1e-2;
    for(int i = 0; i < num_pairs; i++) {
        if(abs(variances_gpu[i] - variances_cpu[i]) > epsilon) {
            printf("Verification failed at index %d: GPU=%f, CPU=%f\n", i, variances_gpu[i], variances_cpu[i]);
            return false;
        }
    }
    return true;
}


PerformanceMetrics benchmarkLogVarianceRatio() {
    PerformanceMetrics metrics;
    
    // Allocate host memory
    float *h_Y = (float*)malloc(nb_samples * nb_genes * sizeof(float));
    float *h_variances_cpu = (float*)malloc(num_pairs * sizeof(float));
    float *h_variances_gpu = (float*)malloc(num_pairs * sizeof(float));
    
    // Initialize matrices
    initializeMatrice(h_Y, nb_samples, nb_genes);
    
    // Allocate device memory
    float *d_Y, *d_variances_gpu;
    hipMalloc(&d_Y, nb_samples * nb_genes * sizeof(float));
    hipMalloc(&d_variances_gpu, num_pairs * sizeof(float));
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Memory transfer timing
    hipEventRecord(start);
    hipMemcpy(d_Y, h_Y, nb_samples * nb_genes * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&metrics.memory_time, start, stop);
    
    hipEventRecord(start);
    computeLogRatioVariance<<<num_blocks, threads_per_block>>>(d_Y, d_variances_gpu, nb_samples, nb_genes);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&metrics.kernel_time, start, stop);
    
    // Copy result back
    hipMemcpy(h_variances_gpu, d_variances_gpu, num_pairs * sizeof(float), hipMemcpyDeviceToHost);
    
    // Calculate performance metrics
    metrics.total_time = metrics.kernel_time + metrics.memory_time;
    float operations = 2.0f * nb_samples * nb_genes * nb_genes;  // multiply-add per element
    metrics.gflops = (operations / 1e9) / (metrics.kernel_time / 1000.0f);
    metrics.bandwidth = (3.0f * nb_samples * nb_genes * sizeof(float)) / (metrics.total_time * 1e6);  // GB/s
    
    // Verify results
    h_variances_cpu = compute_log_variance_ratio_cpu(h_Y, nb_samples, nb_genes);
    bool correct = verifyResults(h_variances_gpu, h_variances_cpu, num_pairs);
    
    // Print performance metrics
    printf("\nPerformance Metrics:\n");
    printf("Matrix Size: %dx%d\n", nb_samples, nb_genes);
    printf("  +-- Kernel Time:     %.2f ms\n", metrics.kernel_time);
    printf("  +-- Memory Time:     %.2f ms\n", metrics.memory_time);
    printf("Total Time: %.3f ms\n", metrics.total_time);
    printf("Performance: %.2f GFLOPs\n", metrics.gflops);
    printf("Memory Bandwidth: %.2f GB/s\n", metrics.bandwidth);
    printf("Results: %s\n", correct ? "PASSED" : "FAILED");
    
    // Cleanup
    free(h_Y); free(h_variances_cpu); free(h_variances_gpu);
    hipFree(d_Y);
    hipFree(d_variances_gpu);
    
    return metrics;
}

int main() {
    // Get device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("\nDevice: %s\n", prop.name);
    printf("Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max threads in X-dimension: %d\n", prop.maxThreadsDim[0]);
    
    // Run benchmark
    PerformanceMetrics metrics = benchmarkLogVarianceRatio();

    // print the metrics to console 
    printf("\n=== Log Variance Ratio Benchmark Report ===\n");
    printf("============================================\n");
    printf("Performance Summary:\n");
    printf("--------------------------------------------\n");
    printf("Total Execution Time: %.2f ms\n", metrics.total_time);
    printf("  +-- Kernel Time:     %.2f ms\n", metrics.kernel_time); 
    printf("  +-- Memory Time:     %.2f ms\n", metrics.memory_time);
    printf("\nCompute Performance:\n");
    printf("--------------------------------------------\n");
    printf("GFLOP/s:             %.2f\n", metrics.gflops);
    printf("Memory Bandwidth:     %.2f GB/s\n", metrics.bandwidth);
    printf("============================================\n");

    // Run CPU benchmark for comparison
    float cpu_time;
    {
        float *h_Y = (float*)malloc(nb_samples * nb_genes * sizeof(float));
        float *h_variances_cpu = (float*)malloc(num_pairs * sizeof(float));
        
        initializeMatrice(h_Y, nb_samples, nb_genes);
        
        auto start_time = clock();
        compute_log_variance_ratio_cpu(h_Y, nb_samples, nb_genes);
        auto end_time = clock();
        
        cpu_time = (float)(end_time - start_time) / CLOCKS_PER_SEC * 1000.0f; // Convert to ms
        
        free(h_Y);
        free(h_variances_cpu);
    }
    
    // Calculate CPU metrics
    float cpu_gflops = (2.0f * nb_samples * nb_genes * nb_genes) / (cpu_time * 1e6);
    
    printf("\n=== CPU vs GPU Comparison ===\n");
    printf("--------------------------------------------\n");
    printf("CPU Time:             %.2f ms\n", cpu_time);
    printf("GPU Time:             %.2f ms\n", metrics.total_time);
    printf("Speedup:              %.2fx\n", cpu_time / metrics.total_time);
    printf("\nCompute Performance:\n");
    printf("CPU GFLOP/s:          %.2f\n", cpu_gflops);
    printf("GPU GFLOP/s:          %.2f\n", metrics.gflops);
    printf("Performance Ratio:     %.2fx\n", metrics.gflops / cpu_gflops);
    printf("============================================\n");
    
    return 0;
}
